#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"
#include "math.h"

#include <stdio.h>

__global__ void sobelCUDA(unsigned char* inputImage, unsigned char* outputImage, int imageWidth, int imageHeight) {

	int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
	int pixelY = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = pixelX + pixelY * imageWidth;

	char hKernel[9] = { -1, 0, 1,
						-2, 0, 2,
						-1, 0, 1 };

	char vKernel[9] = { -1, -2, -1,
						 0,  0,  0,
						 1,  2,  1 };

	if (((pixelX > 0) && (pixelX < imageWidth-1)) && ((pixelY > 0) && (pixelY < imageHeight-1))) {

		// Magnitudes
		int magXR = 0, magXG = 0, magXB = 0;
		int magYR = 0, magYG = 0, magYB = 0;

		// Horizontal and Vertical convolution at point (x,y)
		for (int h = 0; h < 3; h++)
		{
			for (int v = 0; v < 3; v++)
			{
				// Current pixel in 3x3 convolution window
				int xn = pixelX + (h - 1);
				int yn = pixelY + (v - 1);

				int inputPixel = (xn + yn * imageWidth) * 4;
				// Horizontal Convolution
				int hKernelValue = hKernel[h*3 + v];
				magXR += inputImage[inputPixel] * hKernelValue;
				magXG += inputImage[inputPixel+1] * hKernelValue;
				magXB += inputImage[inputPixel+2] * hKernelValue;
				// Vertical Convolution
				int vKernelValue = vKernel[h*3 + v];
				magYR += inputImage[inputPixel] * vKernelValue;
				magYG += inputImage[inputPixel+1] * vKernelValue;
				magYB += inputImage[inputPixel+2] * vKernelValue;
			}
		}

		// Compute final pixel value
		// We clip the value to 255 in case we go over the 8-bit range
		// Instead of using norm, we apprximate using sum of abs, this
		// has been show to be as effective in this application
		int finalR = min(abs(magXR) + abs(magYR), 255);
		int finalG = min(abs(magXG) + abs(magYG), 255);
		int finalB = min(abs(magXB) + abs(magYB), 255);

		outputImage[offset * 4] = finalR;
		outputImage[offset * 4 + 1] = finalG;
		outputImage[offset * 4 + 2] = finalB;
		outputImage[offset * 4 + 3] = 255;
	}
}

void __declspec(dllexport) __cdecl sobelFilterCUDA(unsigned char* inputImageHost, unsigned char* outputImageHost, int imageWidth, int imageHeight) {

	unsigned char* inputImageDevice;
	unsigned char* outputImageDevice;
	int imageSize = imageWidth * imageHeight * 4 * sizeof(unsigned char);

	hipMalloc((void**)&inputImageDevice, imageSize);
	hipMalloc((void**)&outputImageDevice, imageSize);
	hipMemcpy(inputImageDevice, inputImageHost, imageSize, hipMemcpyHostToDevice);

	dim3 blockDims(16, 16);
	dim3 gridDims((unsigned int)ceil(((double)imageWidth  / blockDims.x)),
				  (unsigned int)ceil(((double)imageHeight / blockDims.y)));
	
	sobelCUDA<<<gridDims, blockDims>>>(inputImageDevice, outputImageDevice, imageWidth, imageHeight);

	hipMemcpy(outputImageHost, outputImageDevice, imageSize, hipMemcpyDeviceToHost);
	hipFree(inputImageDevice);
	hipFree(outputImageDevice);
}